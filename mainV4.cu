#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <windows.h>
#include <locale.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define SIGMA_MAX 0.5
#define ROWS_MATRIX 2160
#define COLUMNS_MATRIX 1440
#define ROWS_FILTER 7
#define COLUMNS_FILTER ROWS_FILTER
#define MAX_NUMBER 255
#define MIN_NUMBER 0
#define THREADS_PER_BLOCK 256
#define DEBUG 0

__constant__ float filters[ROWS_FILTER * COLUMNS_FILTER * 255];

__device__ uint8_t applyFilter(const uint8_t* __restrict__ matrix, uint16_t x, uint16_t y, const float* __restrict__ filter) {
    float result = 0;
    uint16_t i, j;

    uint16_t startX = 0;
    uint16_t startY = 0;
    uint16_t HALF_ROW = (ROWS_FILTER / 2);
    uint16_t HALF_COLUMN = (COLUMNS_FILTER / 2);
    if(x < HALF_ROW) startX = HALF_ROW - x;
    if(y < HALF_COLUMN) startY = HALF_COLUMN - y;

    uint16_t endX = ROWS_FILTER;
    uint16_t endY = COLUMNS_FILTER;
    if(x >= ROWS_MATRIX - HALF_ROW) endX = HALF_ROW + ROWS_MATRIX - x;
    if(y >= COLUMNS_MATRIX - HALF_COLUMN) endY = HALF_COLUMN + COLUMNS_MATRIX - y;

    for (i = startX; i < endX; i++) {
        for (j = startY; j < endY; j++) {
            result += matrix[(x - HALF_ROW + i) * COLUMNS_MATRIX + (y - HALF_COLUMN + j)] * filter[i * COLUMNS_FILTER + j];
        }
    }

    if (result > 255)
        return 255;
    return result;
}

/*
    SCHEDULAZIONE:
    la gpu è composta da Streaming Multiprocessor (SM) che eseguono insiemi di 32 threads (warps).
    Al lancio del kernel, uno SM riceve blocksPerGrid blocchi definiti in bidimensionalConvolution<<<blocksPerGrid, threadsPerBlock>>>.
    Un blocco rimane sullo SM fino alla fine dell'elaborazione.
    Il blocco viene diviso in warps da 32 threads. I warps vengono schedulati per l'esecuzione sullo SM.
    Lo SM può arrivare in genere a 64 warps attivi.
    In base al numero di threads che assegnamo per blocco, andiamo a definire il numero di warps usati per blocco.
    Se il numero di threads per blocco non è un multiplo di 32, per ogni blocco si creerà un warp parziale (con meno di 32 threads) che andrà
    a peggiorare l'utilizzazione dell'architettura.
    Pochi warps -> bassa utilizzazione
    Troppi warps -> inefficienza
*/

__global__ void bidimensionalConvolution(const uint8_t* __restrict__ imgs, const uint8_t* __restrict__ blurMap, uint8_t* results, uint16_t nBlocks, uint16_t layersNum) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int totThreads = nBlocks * THREADS_PER_BLOCK;

    if (idx >= totThreads) {
        printf("Thread out of range\n");
        return;
    }

    uint64_t basePixels = (ROWS_MATRIX * COLUMNS_MATRIX * layersNum) / totThreads;
    uint64_t extraPixels = (ROWS_MATRIX * COLUMNS_MATRIX * layersNum) % totThreads;

    uint64_t start, end;

    if (idx < extraPixels) {
        start = idx * (basePixels + 1);
        end = start + basePixels + 1;
    } else {
        start = idx * basePixels + extraPixels;
        end = start + basePixels;
    }

    for(uint64_t j = start; j < end; j++) {
        uint8_t blurValue = blurMap[j % (ROWS_MATRIX * COLUMNS_MATRIX)];
        if(blurValue == 0) {
            results[j] = imgs[j];
            continue;
        }

        uint64_t layer = j / ((uint64_t)ROWS_MATRIX * COLUMNS_MATRIX);
        uint64_t rem   = j % ((uint64_t)ROWS_MATRIX * COLUMNS_MATRIX);
        uint64_t row   = rem / COLUMNS_MATRIX;
        uint64_t col   = rem % COLUMNS_MATRIX;

        results[j] = applyFilter(imgs + layer * ROWS_MATRIX * COLUMNS_MATRIX, row, col, filters + (blurValue - 1) * ROWS_FILTER * COLUMNS_FILTER);
    }
}

void imgsCudaMalloc(uint8_t n, uint16_t rows, uint16_t cols, uint8_t** cudaPointer, uint8_t toFill);
void blurMapCudaMalloc(uint16_t rows, uint16_t cols, uint8_t** cudaPointer);
void precomputeFilters(float** cudaPointer);

uint16_t LAYERS_NUM;
uint8_t* imgs;
uint8_t* blurMap;

float experiment(uint16_t nBlocks) {
    LARGE_INTEGER start, end, freq;
    QueryPerformanceFrequency(&freq);

    // allocazione
    uint8_t* d_imgs = nullptr;        //d_imgs[n_img][i][j] = d_imgs[n_img * (rows * cols) + i * cols + j];
    uint8_t* d_blurMap = nullptr;
    uint8_t* d_results = nullptr;
    imgsCudaMalloc(LAYERS_NUM, ROWS_MATRIX, COLUMNS_MATRIX, &d_imgs, 1);
    blurMapCudaMalloc(ROWS_MATRIX, COLUMNS_MATRIX, &d_blurMap);
    imgsCudaMalloc(LAYERS_NUM, ROWS_MATRIX, COLUMNS_MATRIX, &d_results, 0);

    if(DEBUG) {
        printf("starting %d threads\n", nBlocks * THREADS_PER_BLOCK);
        printf("rows per thread: %d\n", ROWS_MATRIX / (nBlocks * THREADS_PER_BLOCK));
    }

    QueryPerformanceCounter(&start);

    // Lancio del kernel
    bidimensionalConvolution<<<nBlocks, THREADS_PER_BLOCK>>>(d_imgs, d_blurMap, d_results, nBlocks, LAYERS_NUM);

    //controllo errori di lancio
    hipError_t err = hipGetLastError();  // controlla errori di lancio kernel
    if (err != hipSuccess) {
        printf("Errore lancio kernel: %s\n", hipGetErrorString(err));
    }
    hipError_t errSync = hipDeviceSynchronize();
    QueryPerformanceCounter(&end);
    //controllo errori finali
    if (errSync != hipSuccess) {
        printf("Errore runtime kernel: %s\n", hipGetErrorString(errSync));
    }

    float elapsedTime = (float)(end.QuadPart - start.QuadPart) / freq.QuadPart * 1000.0;

    hipFree(d_imgs);
    hipFree(d_blurMap);
    hipFree(d_results);
    return elapsedTime;
}

void concatStringNumber(char *str, int numero) {
    char numStr[20];
    sprintf(numStr, "%d", numero);

    strcat(str, numStr);
}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        printf("./main <N_BLOCKS> <N_IMGS> <saveData>\n");
        return 1; // Esce con codice di errore
    }
    // Converte gli argomenti in interi
    uint16_t NBlocks = atoi(argv[1]);
    uint16_t NImgs = atoi(argv[2]);
    LAYERS_NUM = NImgs;
    uint16_t saveData = atoi(argv[3]);
    uint16_t realNBlocks = NBlocks;

    if(NBlocks * THREADS_PER_BLOCK > ROWS_MATRIX * COLUMNS_MATRIX * LAYERS_NUM) {
        NBlocks = (ROWS_MATRIX * COLUMNS_MATRIX * LAYERS_NUM) / THREADS_PER_BLOCK;
        printf("thread limitati a %d\n", NBlocks);
    }

    float elapsedTime = experiment(NBlocks);
    printf("Elapsed time = %f ms\n", elapsedTime);

    if(!saveData) {
        return 0;
    }

    if(setlocale(LC_NUMERIC, "Italian_Italy.1252") == NULL) {
        printf("Failed to set locale\n");
        return 1;
    }

    char filename[100] = "resultsV4/executionTime_";
    concatStringNumber(filename, NImgs);
    strcat(filename, "IMGS.csv\0");
    FILE* file = fopen(filename, "r");
    int exists = file != NULL;
    if (file != NULL) {
        exists = 1;
        fclose(file);
    }
    file = fopen(filename, "a");

    if(exists == 0) {
        fprintf(file, "Threads;NImgs;RowsFilter;executionTime\n");
    }

    fprintf(file, "%d;%d;%d;%.3f\n", realNBlocks * THREADS_PER_BLOCK, NImgs, ROWS_FILTER, elapsedTime);
    fclose(file);
    return 0;
}

static uint32_t rng_state = 123456789;
uint8_t randomNumber() {
    rng_state = 1664525 * rng_state + 1013904223;
    return (uint8_t)(rng_state >> 24);
}

void fillMatrix(uint8_t* matrix, uint16_t rows, uint16_t cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            matrix[i * cols + j] = randomNumber();
        }
    }
}

void imgsCudaMalloc(uint8_t n, uint16_t rows, uint16_t cols, uint8_t** cudaPointer, uint8_t toFill) {
    uint8_t* img = (uint8_t*)malloc(cols * rows * sizeof(uint8_t));
    int totalSize = n * cols * rows;

    hipError_t err = hipMalloc(cudaPointer, totalSize * sizeof(uint8_t));
    if (err != hipSuccess) {
        printf("hipMalloc failed: %s\n", hipGetErrorString(err));
        exit(1);
    }

    for (int i = 0; i < n; i++) {
        if(toFill > 0) {
            fillMatrix(img, rows, cols);
        }
        hipMemcpy((*cudaPointer) + i * (rows * cols), img, rows * cols * sizeof(uint8_t), hipMemcpyHostToDevice);
    }

    free(img);
}

void disegna_cerchio_sfumato(uint8_t* matrice, int width, int height) {
    int centerX = width / 2;
    int centerY = height / 2;
    float radius = width / 3.0f;

    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            int dx = x - centerX;
            int dy = y - centerY;
            float distanza = sqrtf(dx * dx + dy * dy);

            if (distanza <= radius) {
                float valore = 255.0f * (1.0f - (distanza / radius));
                matrice[y * width + x] = (uint8_t)(valore + 0.5f); // arrotondamento
            } else {
                matrice[y * width + x] = 0;
            }
        }
    }
}

void blurMapCudaMalloc(uint16_t rows, uint16_t cols, uint8_t** cudaPointer) {
    uint8_t* img = (uint8_t*)malloc(cols * rows * sizeof(uint8_t));
    disegna_cerchio_sfumato(img, cols, rows);

    int totalSize = cols * rows;
    hipError_t err = hipMalloc(cudaPointer, totalSize * sizeof(uint8_t));
    if (err != hipSuccess) {
        printf("hipMalloc failed: %s\n", hipGetErrorString(err));
        exit(1);
    }
    hipMemcpy(*cudaPointer, img, totalSize * sizeof(uint8_t), hipMemcpyHostToDevice);
    free(img);
}

// depends on sigma and the coords of the filter
float gaussianBlur(uint16_t i, uint16_t j, float sigma) {
    float denominator = 2.51 * sigma;

    int16_t it = i - ROWS_FILTER / 2;
    int16_t jt = j - COLUMNS_FILTER / 2;

    float exponent = (it * it + jt * jt) / (2 * sigma * sigma);
    return (1.0 / denominator) * exp(-exponent);
}

// to compute the filter given the coords of the matrix
void computeFilter(float* filter, uint8_t blurMapValue) {
    for (uint16_t i = 0; i < ROWS_FILTER; i++) {
        for (uint16_t j = 0; j < COLUMNS_FILTER; j++) {
            filter[i * COLUMNS_FILTER + j] = gaussianBlur(i, j, blurMapValue);
        }
    }
}

void precomputeFilters() {
    float* h_filters = (float*)malloc(ROWS_FILTER * COLUMNS_FILTER * sizeof(float) * 255);

    for (uint16_t i = 0; i < 255; i++) {
        computeFilter(h_filters + i * (ROWS_FILTER * COLUMNS_FILTER), i + 1);
    }

    hipMemcpyToSymbol(HIP_SYMBOL(filters), h_filters, ROWS_FILTER * COLUMNS_FILTER * sizeof(float) * 255);
}